#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

const char* getfield(char* line, int num){
  const char* tok;
  for (tok = strtok(line, ",");
  tok && *tok;
  tok = strtok(NULL, ",\n"))
  {
    if (!--num)
    return tok;
  }
  return NULL;
}

__global__ void calcPot(double *u, double *unext, int *currRefracIter, double *currTime, int *d_it, double *w, double *lastTime, double *d_refracTime,
int *d_maxRefracIter, int *d_N, int *d_R, double *d_uth, int *d_minMPVIter, double *d_dt, double *d_mi, double *d_sumCoeff){

  double pi=3.14159265359;
  int it = *d_it;
  double refracTime = *d_refracTime;
  int maxRefracIter = *d_maxRefracIter;
  int N = *d_N;
  int R = *d_R;
  double uth = *d_uth;
  int minMPVIter = *d_minMPVIter;
  double dt = *d_dt;
  double mi = *d_mi;
  double sumCoeff = *d_sumCoeff;
  int myId = blockDim.x * blockIdx.x + threadIdx.x;

  /*******Refractory Period*******/
  if (*(u+myId)==0 && *(currRefracIter+myId)<maxRefracIter){
    (*(currRefracIter+myId))++;
    return;
  }
  else{
    *(currRefracIter+myId)=0;
  }

  /*******Sum Calculation*******/
  double sumVar=0.0;
  int k,l;
  int iLeftCorner=N+myId/N-R;
  int jLeftCorner=N+myId%N-R;
  for  (k=iLeftCorner; k<iLeftCorner+2*R+1; k++){
    for (l=jLeftCorner; l<jLeftCorner+2*R+1; l++){
      sumVar+=*(u+myId)-*(u+(k%N)*N+l%N);
    }
  }

  *(unext+myId)=*(u+myId)+dt*(mi-*(u+myId)+sumCoeff*sumVar);
  *(currTime+myId)+=dt;

  if(*(unext+myId)>=uth){   //Threshold crossed
    *(unext+myId)=0.0;
    if (it>=minMPVIter){
      *(w+myId)=((*(w+myId))*(*(lastTime+myId))+2*pi)/((*(lastTime+myId))+(*(currTime+myId))+refracTime);
      *(lastTime+myId)+=(*(currTime+myId))+refracTime;
    }
    *(currTime+myId)=0.0;
  }

  return;
}

int main(int argc, char** argv){

  FILE *file1;
  char filename[100];

  /*******Parameter Declarations*******/
  int N=100;  //Grid dimension
  double dt=0.001; //0.001
  int totalTime=10000;  //Simulation time
  int it=0;
  int totalIter=totalTime/dt; //Total iterations
  int R=22; //Square radius
  double sigma=0.7; //Coupling strength
  double sumCoeff=sigma/((2*R+1)*(2*R+1)-1);  //Potential sum coefficient
  double mi=1.0; //Integrator floor
  double uth=0.98;

  double Ts=log(mi/(mi-uth));
  double refracTime=0.22*Ts;  //Refractory period time
  int maxRefracIter=(int)ceil(refracTime/dt); //Refractory period iterations
  int i,j;

  double u[N*N];
  double unext[N*N];

  int currRefracIter[N*N];  //Current iterations already in refractory period
  int maxMPVIter=30000;
  int minMPVIter=2000000; //bhma meta to opoio me endiaferei na arxisw na upologizw thn syxnothta twn neurwnwn.

  double currTime[N*N];
  double lastTime[N*N];
  double w[N*N];
  double t=0.0;

  for (i=0; i<N; i++){
    for (j=0; j<N; j++){
      (*(unext+i*N+j))=0.0;
      (*(currTime+i*N+j))=0.0;
      (*(lastTime+i*N+j))=0.0;
      (*(currRefracIter+i*N+j))=0.0;
    }
  }
  file1=fopen(argv[1],"r"); //argv[1]
  char line[2048];
  i=0;
  while(fgets(line, 2048, file1)){
    for(j=1;j<=N;j++){
      char* tmp = strdup(line);
      (*(u+N*i+j-1))=atof(getfield(tmp,j));
      free(tmp);
    }
    i++;
  }
  fclose(file1);

  double *d_u, *d_unext, *d_currTime, *d_w, *d_lastTime, *d_refracTime, *d_uth, *d_dt, *d_mi, *d_sumCoeff;
  int *d_currRefracIter, *d_it, *d_maxRefracIter, *d_N, *d_R, *d_minMPVIter;

  hipMalloc(&d_u, N*N*sizeof(double));
  hipMalloc(&d_unext, N*N*sizeof(double));
  hipMalloc(&d_currRefracIter, N*N*sizeof(int));
  hipMalloc(&d_currTime, N*N*sizeof(double));
  hipMalloc(&d_it, sizeof(int));
  hipMalloc(&d_w, N*N*sizeof(double));
  hipMalloc(&d_lastTime, N*N*sizeof(double));

  hipMalloc(&d_refracTime, sizeof(double));
  hipMalloc(&d_maxRefracIter, sizeof(int));
  hipMalloc(&d_N, sizeof(int));
  hipMalloc(&d_R, sizeof(int));
  hipMalloc(&d_uth, sizeof(double));
  hipMalloc(&d_minMPVIter, sizeof(int));
  hipMalloc(&d_dt, sizeof(double));
  hipMalloc(&d_mi, sizeof(double));
  hipMalloc(&d_sumCoeff, sizeof(double));

  hipMemcpy(d_refracTime, &refracTime, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_maxRefracIter, &maxRefracIter, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_R, &R, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_uth, &uth, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_minMPVIter, &minMPVIter, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dt, &dt, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_mi, &mi, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_sumCoeff, &sumCoeff, sizeof(double), hipMemcpyHostToDevice);

  time_t benchBegin = time(NULL);
  /*******Simulation*******/
  while (it<totalIter){

    if (it%10000==0) printf("Iteration %d of %d\n", it, totalIter);

    hipMemcpy(d_u, u, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_unext, unext, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_currRefracIter, currRefracIter, N*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_currTime, currTime, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_it, &it, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lastTime, lastTime, N*N*sizeof(double), hipMemcpyHostToDevice);

    //printf("STARTING\n");
    calcPot<<<100,100>>>(d_u, d_unext, d_currRefracIter, d_currTime, d_it, d_w, d_lastTime, d_refracTime, d_maxRefracIter, d_N, d_R, d_uth, d_minMPVIter, d_dt, d_mi, d_sumCoeff);
    hipDeviceSynchronize();

    hipMemcpy(unext, d_unext, N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(currRefracIter, d_currRefracIter, N*N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(currTime, d_currTime, N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(w, d_w, N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(lastTime, d_lastTime, N*N*sizeof(double), hipMemcpyDeviceToHost);

    // for(i=0; i<N; i++){
    //   for(j=0; j<N; j++){
    //     printf("%lf ", *(unext+N*i+j));
    //   }
    //   printf("\n");
    // }
    // printf("FINISHED\n");

    if(it%10000==0){
      sprintf(filename, "ResultsCUDA%s/Results_POT_LIF_2D_Classic_sigma_%lf_R_%d_time_%lf_.dat",argv[2],sigma,R,t);
      file1=fopen(filename,"w");
      for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          fprintf(file1, "%lf,",*(unext+N*i+j));
        }
        fprintf(file1,"\n");
      }
      fclose(file1);
    }
    if (it>minMPVIter){
      if ((it-minMPVIter)%maxMPVIter==0){
        sprintf(filename, "ResultsCUDA%s/Results_MPV_LIF_2D_Classic_sigma_%lf_R_%d_time_%lf_.dat",argv[2],sigma,R,t);
        file1=fopen(filename,"w");
        for(i=0;i<N;i++){
          for(j=0;j<N;j++){
            fprintf(file1,"%lf,",*(w+N*i+j));
          }
          fprintf(file1,"\n");
        }
        fclose(file1);
      }
    }
    if (it == 2000000){
      time_t benchEnd = time(NULL);
      sprintf(filename, "ResultsCUDA%s/execTime.dat",argv[2]);
      file1=fopen(filename,"w");
      fprintf(file1,"Execution time for 2000 time units: %ld seconds\n",benchEnd-benchBegin);
      fclose(file1);
    }
    for (i=0; i<N; i++){
      for (j=0; j<N; j++){
        (*(u+N*i+j))=*(unext+N*i+j);
      }
    }
    t+=dt;
    it++;
  } //edw kleinei h while.

  hipFree(d_u);
  hipFree(d_unext);
  hipFree(d_currRefracIter);
  hipFree(d_currTime);
  hipFree(d_it);
  hipFree(d_w);
  hipFree(d_lastTime);
  hipFree(d_refracTime);
  hipFree(d_maxRefracIter);
  hipFree(d_N);
  hipFree(d_R);
  hipFree(d_uth);
  return(0);
}
